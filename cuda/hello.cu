/**/


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
    # if __CUDA_ARCH__>=200
        printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
    #endif  
}

int main(void){

    // int num_threads = STR_LENGTH;
    // int num_blocks = 2;
    // dim3 dimBlock(16, 16);
    // dim3 dimGrid(32, 32);
    hello<<<1, STR_LENGTH>>>();
    hipDeviceSynchronize();

    return 0;

}
