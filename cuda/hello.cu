/**/


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
        printf("%c\n", STR[threadIdx.x % STR_LENGTH]);

}

int main(void){

    // int num_threads = STR_LENGTH;
    // int num_blocks = 2;
    // dim3 dimBlock(16, 16);
    // dim3 dimGrid(32, 32);
    hello<<<1, STR_LENGTH>>>();
    hipDeviceSynchronize();

    return 0;

}
