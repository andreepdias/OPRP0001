#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

__global__ void matrixMultKernel(double *d_m, double *d_n, double *d_p, int n){

    __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    double pvalue = 0;

    for(int ph = 0; ph < n / TILE_WIDTH; ph++){

        Mds[ty][tx] = d_m[row * n + ph * TILE_WIDTH + tx];
        Nds[ty][tx] = d_n[(ph * TILE_WIDTH + ty) * n + col];
        __syncthreads();
        
        for(int k = 0; k < TILE_WIDTH; k++){
            pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();
    }
    d_p[row * n + col = pvalue];
}


int main(){

    double *a, *b, *c;
    double *d_a, *d_b, *d_c;
    double size = N * N * sizeof(double);

    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    a = (double*)malloc(size);
    b = (double*)malloc(size);
    c = (double*)malloc(size);

    for(int i = 0; i < N * N; i++){
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    matrixMultKernel<<<(int)ceil(N / THREADS), THREADS>>> (d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    printf("c[0] = %lf\n", c[0]);
    printf("c[%d] = %lf\n", N-1, c[N-1]);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}