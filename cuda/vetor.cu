
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1200
#define THREADS 1024

__global__ void vecAdd (int *a, int *b, int *c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < N){
        c[index] = a[index] + b[index];
    }
}

int main(){

    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    for(int i = 0; i < N; i++){
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // vecAdd<<<1, N>>> (d_a, d_b, d_c);
     vecAdd<<<(int)ceil(N/THREADS), THREADS>>> (d_a, d_b, d_c);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    printf("c[0] = %d\n", c[0]);
    printf("c[%d] = %d\n", N-1, c[N-1]);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}