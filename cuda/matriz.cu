
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1200
#define THREADS 1024

__global__ void matrixMultKernel (double *a, double *b, double *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < n) && (col < n)){
        double v = 0;
        for(int k = 0; k < n; k++){
            v += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = v;
    }
}

int main(){

    double *a, *b, *c;
    double *d_a, *d_b, *d_c;
    double size = N * N * sizeof(double);

    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    a = (double*)malloc(size);
    b = (double*)malloc(size);
    c = (double*)malloc(size);

    for(int i = 0; i < N * N; i++){
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    matrixMultKernel<<<(int)ceil(N / THREADS), THREADS>>> (d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    printf("c[0] = %lf\n", c[0]);
    printf("c[%d] = %lf\n", N-1, c[N-1]);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}